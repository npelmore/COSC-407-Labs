#include "hip/hip_runtime.h"
%%cuda --name convolution.cu --compile true

#include "EasyBMP.h"
#include "EasyBMP.cu"
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>	// for uchar4 struct

#define MIN(x,y) (  (y) ^ (((x) ^ (y)) & -((x) < (y))) )
#define MAX(x,y) (  (x) ^ (((x) ^ (y)) & -((x) < (y))) )
#define CHK(call) { hipError_t err = call; if (err != hipSuccess) { printf("Error%d: %s:%d\n",err,__FILE__,__LINE__); printf(hipGetErrorString(err)); hipDeviceReset(); exit(1); }}


//****************************************************************************************************************
// PARALLEL FUNCTIONS
//****************************************************************************************************************
	/*
	TODO: 	Provide CUDA implementation for parallelizing the two SERIAL functions: convolution_8bits and convolution_32Bits
			Make sure to check for errors from CUDA API calls and from Kernel Launch. 
			Also, time your parallel code and compute the speed-up.
	*/
__device__ void convolution_8bits_parallel(const unsigned char* const image_in, unsigned char* const image_out, const int height, const int width, const float *filter, const int filter_width){
	//only filters with width = odd_number are allowed
	if (filter_width % 2 == 0){
		//you don't have to print anything when running on kernel. 
		printf("Filters with even width are not supported yet. Program terminated!\n");
		//exit(1);
	}
	//Apply the filter to every image pixel (col, row) 
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;
	if(row<height && col<width){
        float sum = 0.0f;
        //Having the filter centered at image pixel (col,row), multiply every filter pixel by the corresponding image pixel, and find the sum
        for (int row_f = -filter_width / 2; row_f <= filter_width / 2; ++row_f)
            for (int col_f = -filter_width / 2; col_f <= filter_width / 2; ++col_f) {
                //get the value of the image pixel for the current filter pixel. If out of boundary, use boundary pixels
                int row_i = MIN(MAX(row + row_f, 0), (height - 1));
                int col_i = MIN(MAX(col + col_f, 0), (width - 1));
                float pxl_image = image_in[row_i * width + col_i];
                //get the value for the current filter 
                float pxl_filter = filter[(row_f + filter_width / 2) * filter_width + col_f + filter_width / 2];
                //multiply image_pixel by filter_pixel and add to final value of the image pixel
                sum += pxl_image * pxl_filter;
            }
        //final value of image pixel(col, row) = the sum of every filter pixel multiplied by the corresponding image pixels around image pixel(col,row)
        image_out[row * width + col] = sum;
	}
    
}

//	This function applies the convolution kernel (denoted by filter) to every pixel of the input image (image_in)
//	Constraints:- Both image_in and image_out are in RGBA format (32-bit pixels as uchar4)
//				- Filter is a square matrix (float) and its width is odd number. The sum of all its values is 1 (normalized)

__global__ void convolution_32bits_parallel(const uchar4* const image_in, uchar4 *const image_out, int height, int width, const float* const filter, const int filter_width, unsigned char* R_in, unsigned char* G_in, unsigned char* B_in, unsigned char* A_in, unsigned char* R_out, unsigned char* G_out, unsigned char* B_out, unsigned char* A_out){
	//break the input image (uchar4 matrix) into 4 channels (four char matrices): Red, Green, Blue, and Alpha

    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;

    int i = row*width + col;
        
	//perform 8-bit convolution for each 8-bit image channel 
	convolution_8bits_parallel(R_in, R_out, height, width, filter, filter_width);
	convolution_8bits_parallel(G_in, G_out, height, width, filter, filter_width);
	convolution_8bits_parallel(B_in, B_out, height, width, filter, filter_width);
	convolution_8bits_parallel(A_in, A_out, height, width, filter, filter_width);

	//merge the four channels into one output image of type uchar4
	if (i < width*height)
		image_out[i] = make_uchar4(R_out[i], G_out[i], B_out[i], A_out[i]);	
}

__global__ void rgba_initialize(const uchar4* const image_in, uchar4 *const image_out, int height, int width, const float* const filter, const int filter_width, unsigned char* R_in, unsigned char* G_in, unsigned char* B_in, unsigned char* A_in, unsigned char* R_out, unsigned char* G_out, unsigned char* B_out, unsigned char* A_out){

	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	int i = row*width + col;
	
	if (i < width*height){
		uchar4 pxl = image_in[i];
		R_in[i] = pxl.x;
		G_in[i] = pxl.y;
		B_in[i] = pxl.z;
		A_in[i] = pxl.w;
		
	}
}

//****************************************************************************************************************
// SERIAL FUNCTIONS
//****************************************************************************************************************

//	This function applies the convolution kernel (denoted by filter) to every pixel of the input image (image_in)
//	constraints: image_in and image_out have 8-bit pixels (e.g., grayscale images, only one color channel, etc)

void convolution_8bits(const unsigned char* const image_in, unsigned char* const image_out, const int height, const int width, const float *filter, const int filter_width){
	//only filters with width = odd_number are allowed
	if (filter_width % 2 == 0){
		//you don't have to print anything when running on kernel. 
		fprintf(stderr,"Filters with even width are not supported yet. Program terminated!\n");
		exit(1);
	}
	//Apply the filter to every image pixel (col, row) 
	for (int row = 0; row < height; ++row) 
		for (int col = 0; col < width; ++col) {
			float sum = 0.0f;
			//Having the filter centered at image pixel (col,row), multiply every filter pixel by the corresponding image pixel, and find the sum
			for (int row_f = -filter_width / 2; row_f <= filter_width / 2; ++row_f)
				for (int col_f = -filter_width / 2; col_f <= filter_width / 2; ++col_f) {
					//get the value of the image pixel for the current filter pixel. If out of boundary, use boundary pixels
					int row_i = MIN(MAX(row + row_f, 0), (height - 1));
					int col_i = MIN(MAX(col + col_f, 0), (width - 1));
					float pxl_image = image_in[row_i * width + col_i];
					//get the value for the current filter 
					float pxl_filter = filter[(row_f + filter_width / 2) * filter_width + col_f + filter_width / 2];
					//multiply image_pixel by filter_pixel and add to final value of the image pixel
					sum += pxl_image * pxl_filter;
				}
			//final value of image pixel(col, row) = the sum of every filter pixel multiplied by the corresponding image pixels around image pixel(col,row)
			image_out[row * width + col] = sum;
		}
}

//	This function applies the convolution kernel (denoted by filter) to every pixel of the input image (image_in)
//	Constraints:- Both image_in and image_out are in RGBA format (32-bit pixels as uchar4)
//				- Filter is a square matrix (float) and its width is odd number. The sum of all its values is 1 (normalized)

void convolution_32bits( const uchar4* const image_in, uchar4 *const image_out, int height, int width, const float* const filter, const int filter_width){
	//break the input image (uchar4 matrix) into 4 channels (four char matrices): Red, Green, Blue, and Alpha
	unsigned char *R_in = new unsigned char[width * height];
	unsigned char *G_in = new unsigned char[width * height];
	unsigned char *B_in = new unsigned char[width * height];
	unsigned char *A_in = new unsigned char[width * height];
	for (int i = 0; i < width * height; ++i) {	//break each pixel in input image
		uchar4 pxl = image_in[i];
		R_in[i] = pxl.x;
		G_in[i] = pxl.y;
		B_in[i] = pxl.z;
		A_in[i] = pxl.w;
	}

	//perform 8-bit convolution for each 8-bit image channel 
	unsigned char *R_out = new unsigned char[width * height];
	convolution_8bits(R_in, R_out, height, width, filter, filter_width);

	unsigned char *G_out = new unsigned char[width * height];
	convolution_8bits(G_in, G_out, height, width, filter, filter_width);

	unsigned char *B_out = new unsigned char[width * height];
	convolution_8bits(B_in, B_out, height, width, filter, filter_width);

	unsigned char *A_out = new unsigned char[width * height];
	convolution_8bits(A_in, A_out, height, width, filter, filter_width);

	//merge the four channels into one output image of type uchar4
	for (size_t i = 0; i < height * width; ++i) 
		image_out[i] = make_uchar4(R_out[i], G_out[i], B_out[i], A_out[i]);	
}

//**************************************************************
//No need to parallelize any of the functions below this comment
//**************************************************************

//This function reads a BMP image using the EasyBMP library and returns a 1D array representing the RGBA values of the image pixels
//image_out->x is Red, image_out->y is Green, image_out->z is Blue, image_out->w is Alpha
//how to use:	1- in the calling function, declare these variables:	uchar4* img = NULL;	int width = 0, height = 0;
//				2- then call this function								readBMP(filename, &img, &width, &height); 
void readBMP(const char* FileName, uchar4 **image_out, int* width, int* height){
	BMP img;
	img.ReadFromFile(FileName);
	*width = img.TellWidth();
	*height = img.TellHeight();
	uchar4 *const img_uchar4 = (uchar4*)malloc(*width * *height * sizeof(int));
	// save each pixel to image_out as uchar4 in row-major format
	for (int row = 0; row <*height; row++)
		for (int col = 0; col < *width; col++)
			img_uchar4[col + row * *width] = make_uchar4(img(col, row)->Red, img(col, row)->Green, img(col, row)->Blue, img(col, row)->Alpha);	//use row-major
	*image_out = img_uchar4;
}

//This function writes a BMP image using the EasyBMP library
//how to use: in the calling function, call		writeBMP(destination_filename, source_image_array, width, height); 
void writeBMP(const char* FileName, uchar4 *image, int width, int height){
	BMP output;
	output.SetSize(width, height);
	output.SetBitDepth(24);
	// save each pixel to the output image
	for (int row = 0; row < height; row++){		//for each row
		for (int col = 0; col <  width; col++){	//for each col
			uchar4 rgba = image[col + row * width];
			output(col, row)->Red = rgba.x;
			output(col, row)->Green = rgba.y;
			output(col, row)->Blue = rgba.z;
			output(col, row)->Alpha = rgba.w;
		}
	}
	output.WriteToFile(FileName);

}

//Normalize image filter (sum of all values should be 1) 
// the filter is a 2D float array
void normalizeFilter(float* filter, int width){
	//find the sum
	float sum = 0;
	for (int i = 0; i < width*width; i++)
		sum += filter[i];
	//normalize
	for (int i = 0; i < width*width; i++)
		filter[i] /= sum;
}

//this Function reads the convolution-filter image 
//Contrasting: Filter is 32 bit RGPA image. The filter must be sqaure. Filter width must be an odd number 
float* readFilter(const char* filter_image_name, int* filter_width){
	int filterHeight;	//for testing that height = width
	//read filter image as 32 bit RGPA bitmap and check the constraints (square, odd width)
	uchar4* filterImageUchar;
	readBMP(filter_image_name, &filterImageUchar, filter_width, &filterHeight);
	if (*filter_width != filterHeight || *filter_width % 2 == 0){
		fprintf(stderr, "Non-square filters or filters with even width are not supported yet. Program terminated!\n");
		exit(1);
	}
	//convert every pixel to a float number representing its grayscale intensity. Formula used is 0.21 R + 0.72 G + 0.07 B
	float* filter = (float*)malloc(*filter_width * *filter_width * sizeof(float));
	for (int i = 0; i < *filter_width * *filter_width; i++){
		uchar4 element = filterImageUchar[i];
		filter[i] = 0.21 * element.x + 0.72 * element.y + 0.07 * element.z; 
	}
	//Normalization makes sure that the sum of all values in the filter is 1 
	normalizeFilter(filter, *filter_width);
	//return result
	return filter;
}

void checkForGPU() {
    // This code attempts to check if a GPU has been allocated
    // Colab notebooks technically have access to NVCC and will compile and
    // execute CPU/Host code, however, GPU/Device code will silently fail.
    // To prevent such situations, this code will warn the user.
    int count;
    hipGetDeviceCount(&count);
    if (count <= 0 || count > 100) {
        printf("!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
        printf("->WARNING<-: NO GPU DETECTED ON THIS COLLABORATE INSTANCE.\n");
        printf("IF YOU ARE ATTEMPTING TO RUN GPU-BASED CUDA CODE, YOU SHOULD CHANGE THE RUNTIME TYPE!\n");
        printf("!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    }
}

void serial(){
	int filter_width;
	const char* filter_image_name = "./src/filter_blur_21.bmp";	//filter width = 21 pixels
	const char* image_in_name = "./src/okanagan.bmp";
	const char* image_out_name = "okanagan_blur.bmp";

	//load filter
	float* filter = readFilter(filter_image_name, &filter_width);
	printf("Filter loaded...\n");

	//load input image
	int width, height;
	uchar4* image_in;
	readBMP(image_in_name, &image_in, &width, &height);	//image_in will have all pixel information, each pixel as uchar4
	printf("Input image loaded...\n");

	//apply convolution filter to input image
	uchar4* image_out = (uchar4*)malloc(width*height*sizeof(uchar4));	//reserve space in the memory for the output image
	printf("Applying the convolution filter...\n");
	int t = clock();
	convolution_32bits(image_in, image_out, height, width, filter, filter_width);	//filter applied to image_in, results saved in image_out
	t = (clock() - t) * 1000 / CLOCKS_PER_SEC;
	printf("Convolution filter applied. Time taken: %d.%d seconds\n", t / 1000, t % 1000);
	
	//save results to output image
	writeBMP(image_out_name, image_out, width, height);
	printf("Output image saved.\nProgram finished!\n");
}

void parallel(){
	//launch your cuda kernel from here
    int filter_width;
	const char* filter_image_name = "./src/filter_blur_21.bmp";	//filter width = 21 pixels
	const char* image_in_name = "./src/okanagan.bmp";
	const char* image_out_name = "okanagan_blur.bmp";

	//load filter
	float* filter = readFilter(filter_image_name, &filter_width);
	printf("Filter loaded...\n");

    float* device_filter;
    CHK( hipMalloc(&device_filter, filter_width * filter_width * sizeof(float)) );
    CHK( hipMemcpy(device_filter, filter, filter_width * filter_width * sizeof(float), hipMemcpyHostToDevice) );

	//load input image
	int width, height;
	uchar4* image_in;
	readBMP(image_in_name, &image_in, &width, &height);	//image_in will have all pixel information, each pixel as uchar4
	printf("Input image loaded...\n");

    uchar4* device_image_in;
    CHK( hipMalloc(&device_image_in, width * height * sizeof(uchar4)) );
    CHK( hipMemcpy(device_image_in, image_in, width*height*sizeof(uchar4), hipMemcpyHostToDevice) ); 

	//apply convolution filter to input image
	uchar4* image_out = (uchar4*)malloc(width*height*sizeof(uchar4));	//reserve space in the memory for the output image
	uchar4* device_image_out;
    CHK( hipMalloc(&device_image_out, width * height * sizeof(uchar4)) );

    unsigned char* dR_in;
    CHK( hipMalloc(&dR_in, width * height) );
    unsigned char* dG_in;
    CHK( hipMalloc(&dG_in, width * height) );
    unsigned char* dB_in;
    CHK( hipMalloc(&dB_in, width * height) );
    unsigned char* dA_in;
    CHK( hipMalloc(&dA_in, width * height) );

    unsigned char* dR_out;
    CHK( hipMalloc(&dR_out, width * height) );
    unsigned char* dG_out;
    CHK( hipMalloc(&dG_out, width * height) );
    unsigned char* dB_out;
    CHK( hipMalloc(&dB_out, width * height) );
    unsigned char* dA_out;
    CHK( hipMalloc(&dA_out, width * height) );

    dim3 blockSize(32,32);
    int nBlocks_x = (width -1) / 32 +1;
    int nBlocks_y = (height -1) / 32 +1;

    dim3 gridSize(nBlocks_x, nBlocks_y);


    printf("Applying the convolution filter...\n");
	int t = clock();
	rgba_initialize<<<gridSize,blockSize>>>(device_image_in, device_image_out, height, width, device_filter, filter_width, dR_in, dG_in, dB_in, dA_in, dR_out, dG_out, dB_out, dA_out);
	CHK(hipGetLastError()); 
	CHK(hipDeviceSynchronize());

	convolution_32bits_parallel<<<gridSize,blockSize>>>(device_image_in, device_image_out, height, width, device_filter, filter_width, dR_in, dG_in, dB_in, dA_in, dR_out, dG_out, dB_out, dA_out);	//filter applied to image_in, results saved in image_out
	CHK(hipGetLastError()); 
	CHK(hipDeviceSynchronize());

	t = (clock() - t) * 1000 / CLOCKS_PER_SEC;
	
	printf("Convolution filter applied. Time taken: %d.%d seconds\n", t / 1000, t % 1000);

    CHK( hipMemcpy(image_out, device_image_out, width*height*sizeof(uchar4), hipMemcpyDeviceToHost) ); 
	
	//save results to output image
	writeBMP(image_out_name, image_out, width, height);
	printf("Output image saved.\nProgram finished!\n");

	hipFree(device_filter); 
	hipFree(device_image_in);
	hipFree(device_image_out);
	hipFree(dR_in);
	hipFree(dG_in);
	hipFree(dB_in);
	hipFree(dA_in);
	hipFree(dR_out);
	hipFree(dG_out);
	hipFree(dB_out);
	hipFree(dA_out);
}

//MAIN: testing convolution with a blur filter
int main(){
  checkForGPU();
	serial();
	parallel();
}


// Filter loaded...
// Input image loaded...
// Applying the convolution filter...
// Convolution filter applied. Time taken: 11.993 seconds
// Output image saved.
// Program finished!
// Filter loaded...
// Input image loaded...
// Applying the convolution filter...
// Convolution filter applied. Time taken: 0.7 seconds
// Output image saved.
// Program finished!