#include "hip/hip_runtime.h"
%%cu

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include ""

#define CHK(call) { hipError_t err = call; if (err != hipSuccess) { printf("Error%d: %s:%d\n",err,__FILE__,__LINE__); printf(hipGetErrorString(err)); hipDeviceReset(); exit(1); }}

void checkForGPU() {
    // This code attempts to check if a GPU has been allocated.
    // Colab notebooks without a GPU technically have access to NVCC and will
    // compile and execute CPU/Host code, however, GPU/Device code will silently
    // fail. To prevent such situations, this code will warn the user.
    int count;
    hipGetDeviceCount(&count);
    if (count <= 0 || count > 100) {
        printf("!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
        printf("->WARNING<-: NO GPU DETECTED ON THIS COLLABORATE INSTANCE.\n");
        printf("IF YOU ARE ATTEMPTING TO RUN GPU-BASED CUDA CODE, YOU SHOULD CHANGE THE RUNTIME TYPE!\n");
        printf("!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    }
}

__global__ void reduction1(float* arr, float* partialSums){
    __shared__ float partialSum[512];

    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.x;
    partialSum[i] = arr[x];
    __syncthreads();
    
    for (int stride = 1; stride<blockDim.x; stride *= 2){
    //for (int stride = 1; stride<blockDim.x; stride <<= 1){ // reduction with << instead of *
        if (i % (2 * stride) == 0)
            partialSum[i] += partialSum[i + stride];
        __syncthreads();
    }
    if(i==0)
        partialSums[blockIdx.x] += partialSum[0];
}

__global__ void reduction2(float* arr, float* partialSums){
    __shared__ float partialSum[512];

    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.x;
    partialSum[i] = arr[x];
    __syncthreads();
    
    for (int stride = blockDim.x/2; stride >= 1; stride = stride/2){
        if (i<stride)
            partialSum[i] += partialSum[i + stride];
        __syncthreads();
    }
    if(i==0)
        partialSums[blockIdx.x] += partialSum[0];
}

__global__ void reduction3(float* arr){
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.x;

    for (int stride = 1; stride<blockDim.x; stride *= 2){
        if (i % (2 * stride) == 0)
            arr[x] += arr[x + stride];
        __syncthreads();
    }
}

__global__ void reduction4(float* arr){
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.x;

    for (int stride = blockDim.x/2; stride >= 1; stride = stride/2){
        if (i < stride)
            arr[x] += arr[x + stride];
        __syncthreads();
    }
}

int main() {
    checkForGPU();
    
    const int size = 1<<24;
    int nbytes = size * sizeof(float);
    float* mainArr = (float*) malloc(nbytes);
    float* arr = (float*) malloc(nbytes);
    float* d_arr;
    
    CHK(hipMalloc(&d_arr, nbytes));
    
    srand(time(NULL));
    for(int i=0; i<size; i++)
        //mainArr[i] = 1;
        mainArr[i] = rand() % 256;

    for(int i=0; i<size; i++)
        arr[i] = mainArr[i];

    CHK(hipMemcpy(d_arr, arr, nbytes, hipMemcpyHostToDevice));

    int nthreads = 512;
    int nblocks = (size-1)/nthreads + 1;
    float sum = 0;
    dim3 gridSize(nblocks,1,1);
    dim3 blockSize(nthreads,1,1);
    float* partialSums;
   

    printf("Reducing an array of %d floats on a grid of (%d,1,1) blocks, each block with (%d,1,1) threads\n", size, nblocks, nthreads);

    //reduction 1------------------------------------------------------------------------------------------------------------------------
    CHK(hipMallocManaged(&partialSums, nblocks));
   
    float t = clock();
    reduction1<<<gridSize,blockSize>>>(d_arr, partialSums);
    CHK(hipDeviceSynchronize());
    t = (clock() - t) * 1000 / CLOCKS_PER_SEC;
 
    for (int i=0; i<nblocks; i++){
        sum += partialSums[i];
    }
    
    printf("Using shared memory, More divergence: GPU time: %.3f ms GPU sum: %.2f\n", t, sum);
    
    hipFree(partialSums);
    sum = 0;
    
    //reduction 2------------------------------------------------------------------------------------------------------------------------
    CHK(hipMallocManaged(&partialSums, nblocks));

    t = clock();
    reduction2<<<gridSize,blockSize>>>(d_arr, partialSums);
    CHK(hipDeviceSynchronize());
    t = (clock() - t) * 1000 / CLOCKS_PER_SEC;

    for (int i=0; i<nblocks; i++){
        sum += partialSums[i];
    }
    
    printf("Using shared memory, Less divergence: GPU time: %.3f ms GPU sum: %.2f\n", t, sum);
    
    hipFree(partialSums);
    sum = 0;

    //reduction 3------------------------------------------------------------------------------------------------------------------------
    t = clock();
    reduction3<<<gridSize,blockSize>>>(d_arr);
    CHK(hipDeviceSynchronize());
    t = (clock() - t) * 1000 / CLOCKS_PER_SEC;

    CHK(hipMemcpy(arr, d_arr, nbytes, hipMemcpyDeviceToHost));
    for (int i=0; i<size; i+=size/nblocks){
        sum += arr[i];
    }

    printf("Using global memory, More divergence: GPU time: %.3f ms GPU sum: %.2f\n", t, sum);
   
    for(int i=0; i<size; i++)
        arr[i] = mainArr[i];

    CHK(hipMemcpy(d_arr, arr, nbytes, hipMemcpyHostToDevice));
    sum = 0;

    //reduction 4------------------------------------------------------------------------------------------------------------------------
    t = clock();
    reduction4<<<gridSize,blockSize>>>(d_arr);
    CHK(hipDeviceSynchronize());
    t = (clock() - t) * 1000 / CLOCKS_PER_SEC;

    CHK(hipMemcpy(arr, d_arr, nbytes, hipMemcpyDeviceToHost));
    for (int i=0; i<size; i+=size/nblocks){
        sum += arr[i];
    }

    printf("Using global memory, Less divergence: GPU time: %.3f ms GPU sum: %.2f\n", t, sum);

    //Free malloced and cudaMalloced variables------------------------------------------------------------------------------------------
    free(mainArr);    
    free(arr);    
    hipFree(d_arr);
    
    return 0;
}

/*
Output:
Reducing an array of 16777216 floats on a grid of (32768,1,1) blocks, each block with (512,1,1) threads
Using shared memory, More divergence: GPU time: 3.813 ms GPU sum: 2139051520.00
Using shared memory, Less divergence: GPU time: 2.186 ms GPU sum: 2139051520.00
Using global memory, More divergence: GPU time: 3.423 ms GPU sum: 2139051520.00
Using global memory, Less divergence: GPU time: 1.651 ms GPU sum: 2139051520.00

Difference between * and << on reduction1:
* GPU time: 3.813 ms
<< GPU time: 3.719 ms
Difference: << is 0.094ms faster than *
*/